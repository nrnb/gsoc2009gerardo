#include "hip/hip_runtime.h"
/**************************************************************************************
Copyright (C) Apeksha Godiyal, 2008
Copyright (C) Gerardo Huck, 2009


This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.

See license.h for more information.
**************************************************************************************/

#include "license.h"




#include "kdNode.h"
#include "partition.cu"
#include "cudpp.h"
#include "complexDevice.h"
#include "common.h"


#include <stdio.h>


complexDevice * centerD;

void kdNodeInitD(kdNodeInt * n,kdNodeFloat * nf,int i,unsigned int a,float d,float u,float l,float r){
  int k;
  (n+i-1)->i=i;
  (n+i-1)->numElems=0;
  (n+i-1)->axis=(a);
  (nf+i-1)->up=(u);
  (nf+i-1)->down=(d);
  (nf+i-1)->left=(l);
  (nf+i-1)->right=(r);
  (nf+i-1)->center = Complex((l+r)/2,(u+d)/2); 
  (n+i-1)->child=0;
  (n+i-1)->next=0;
  (nf+i-1)->radius = sqrt(((nf+i-1)->left-(nf+i-1)->center.r)*((nf+i-1)->left-(nf+i-1)->center.r) + ((nf+i-1)->right-(nf+i-1)->center.i)*((nf+i-1)->right-(nf+i-1)->center.i));
	
  for(k = 0; k < 4; k++){
    (nf+i-1)->Outer[k]=Complex(0,0);
  }
}

void printD(kdNodeInt * n, kdNodeFloat * nf,int i){
  if(!n)
    return;
  if(!nf)
    return;
	
  if(((n+i-1)->child==0)){
    printf("%d, %f, %f, %f %f %f\n",(n+i-1)->numElems,(nf+i-1)->down,(nf+i-1)->up,(nf+i-1)->left,(nf+i-1)->right,(nf+i-1)->radius);
    printf("(");
    for(int j = 0; j < (n+i-1)->numElems; j++)
      printf("%d ",(n+i-1)->graphNodes[j]);
    printf(")\n");
    if((n+i-1)->next)
      printD(n,nf,(n+i-1)->next);
  }
  else{
    if((n+i-1)->child)
      printD(n,nf,(n+i-1)->child);
  }
}

void insertListD(kdNodeInt * n,int x,float * NodePos,float pos, int index,int limit){
  int i,j;
  
  for( i = 0; (i < (n+x-1)->numElems) && ((NodePos[2*(n+x-1)->graphNodes[i]+(n+x-1)->axis]) <= pos); i++)
    ;
  for(j = (n+x-1)->numElems; j > i; j--){
    (n+x-1)->graphNodes[j] = (n+x-1)->graphNodes[j-1];
  }
  (n+x-1)->graphNodes[i] = index; 
}

void updateElemsD(kdNodeInt * n, int i, int* start, int* end){
  int j;
  for(j=0;start <=end; start ++,j++){
    (n+i-1)->graphNodes[j] = *start;
  }
  (n+i-1)->numElems = j;
}


void InsertElemD(kdNodeInt * n, kdNodeFloat * nf,int i,float * NodePos,float x,float y, int index, int limit, int init){
  static int nIntAvail=2;
  if(init)
    nIntAvail=2;
  int k;
  float pos;
  
  fcomplex zi = Complex((x),(y));
  fcomplex z_v_minus_z_0_over_k = Csub(zi,((nf+i-1)->center));
  for(k = 0; k < 4; k++){
    fcomplex temp = Complex(-1/(float)(k+1),0);
    (nf+i-1)->Outer[k] = Cadd((nf+i-1)->Outer[k], Cmul(z_v_minus_z_0_over_k,temp));
    z_v_minus_z_0_over_k = Cmul(z_v_minus_z_0_over_k,Csub(zi,(nf+i-1)->center));
    //Outer[k]+=  ((float(-1))*z_v_minus_z_0_over_k)/float(k+1);
    //z_v_minus_z_0_over_k *= zi - (nf+i-1)->center;
  }
  
  if((n+i-1)->axis)
    pos = y;
  else
    pos = x;
  
  if((n+i-1)->child==0){
    insertListD((n),i,NodePos,pos,index,limit);
    if((n+i-1)->numElems >= limit){
      float u1,d1,l1,r1,u2,d2,l2,r2;
      int numElems = (n+i-1)->numElems+1;
      int splitting_node = (n+i-1)->graphNodes[((numElems)/2)-1];
      (nf+i-1)->split = NodePos[2*splitting_node+(n+i-1)->axis];
      if((n+i-1)->axis){
	l1 =(nf+i-1)->left;  l2=(nf+i-1)->left;
	r1 =(nf+i-1)->right; r2 = (nf+i-1)->right;
	u1 =(nf+i-1)->split; u2 = (nf+i-1)->up;
	d1 =(nf+i-1)->down; d2 = (nf+i-1)->split;
      }
      else{
	l1 =(nf+i-1)->left; l2=(nf+i-1)->split;
	r1 =(nf+i-1)->split; r2 = (nf+i-1)->right;
	u1 = (nf+i-1)->up; u2 = (nf+i-1)->up;
	d1 = (nf+i-1)->down; d2 = (nf+i-1)->down;
      }
      (n+i-1)->child = nIntAvail; kdNodeInitD(n,nf,nIntAvail,1-(n+i-1)->axis,d1,u1,l1,r1);
      nIntAvail++;
      (n+nIntAvail-2)->next = nIntAvail; kdNodeInitD(n,nf,nIntAvail,1-(n+i-1)->axis,d2,u2,l2,r2);
      updateElemsD(n,nIntAvail-1,&((n+i-1)->graphNodes[0]),&((n+i-1)->graphNodes[(numElems/2)-1]));
      updateElemsD(n,nIntAvail,&((n+i-1)->graphNodes[(numElems/2)]),&((n+i-1)->graphNodes[limit]));
      (n+nIntAvail-1)->next = (n+i-1)->next;
      nIntAvail++;
    }
  }
  else {
    int c = (n+i-1)->child;
    if(pos <= (nf+i-1)->split)
      InsertElemD((n),nf,c,NodePos,x,y,index,limit,0);
    else
      InsertElemD((n),nf,(n+c-1)->next,NodePos,x,y,index,limit,0);
  }
  (n+i-1)->numElems++;
}

void swapD(float3 * greater,float3 * less){
  float3 temp = *greater;
  *greater = *less;
  *less    = temp;
}

float3 * partition(float3 * a, float3 * b, float pivot,int axis){
  
  int i = 0;
  int n = b-a+1;
  if(axis){
    while((i<n) && ((b-i)->y > pivot))
      i++;
    float3 * greater = b-i;
    i = 0;
    while((i<n) && ((a+i)->y <= pivot))
      i++;
    float3 * less = a+i; 
    for(; less < greater; ){
      swapD(greater,less);
      greater--;
      less++;
      while(less->y <= pivot)
	less++;
      while(greater->y > pivot)
	greater--;
    }
    return greater;
  }
  else{
    while((i<n) && ((b-i)->x > pivot))
      i++;
    float3 * greater = b-i;
    i = 0;
    while((i<n) && ((a+i)->x <= pivot))
      i++;
    float3 * less = a+i; 
    for(; less < greater; ){
      swapD(greater,less);
      greater--;
      less++;
      while(less->x <= pivot)
	less++;
      while(greater->x > pivot)
	greater--;
    }
    return greater;
  }
}

float3 * medianD(float3 * a, float3 * b, int axis,int k){
  
  if(!(a-b))
    return a;
  
  float pivot;
  if(axis){
    pivot = (a)->y;
  }
  else{
    pivot = (a)->x;
  }
  
  float3 * p = partition(a,b,pivot,axis);
  int i = int(p-a)+1;
  if(i==k)
    return p;
  else if(i < k)
    return medianD(p+1,b,axis,k-i);
  else
    return medianD(a,p-1,axis,k);
}

float3 * radixPartitionD(float3 * a, float3 * b, unsigned int bit, int axis){
  // y axis
  unsigned int o = 1;
  o = o<<(bit);
  int n = (int)(b-a)+1;
  if(axis){
    int i=0;
    
    while((i<n) && (((unsigned int)(b-i)->y) & o) )
      i++;
    float3 * greater = b-i;
    if(i==n){
      return greater;
    }
    i = 0;
    while((i<n) && !(((unsigned int)(a+i)->y)&o))
      i++;
    float3 * less = a+i; 
    if(i==n){
      return b;
      //less--;
    }
    for(; less < greater; ){
      swapD(greater,less);
      greater--;
      less++;
      while(!((unsigned int)(less->y)&o))
	less++;
      while(((unsigned int)(greater->y)&o))
	greater--;
    }
    return greater;
  }
  // x axis
  else{
    int i=0;
    while((i<n) && (((unsigned int)(b-i)->x)&o) )
      i++;
    float3 * greater = b-i;
    if(i==n)
      return greater;
    i = 0;
    while((i<n) && !(((unsigned int)(a+i)->x)&o))
      i++;
    float3 * less = a+i;
    if(i==n)
      return b;
    for( ; less < greater ; ){
      swapD(greater,less);
      greater--;
      less++;
      while(!((unsigned int)(less->x)&o))
	less++;
      while(((unsigned int)(greater->x)&o))
	greater--;
    }
    return greater;
  }
}

float3 * radixMedianD(float3* a, float3* b, int axis,int k, unsigned int bit,unsigned int* data_out, unsigned int* d_temp_addr_uint, 
		      float3* d_out,CUDPPHandle planHandle, unsigned int* nD)
{
  float3 * s;
  
  if(a==b)
    return a;
  else{
    int n = b - a + 1; 
    dim3  threads,blocks;
    unsigned m_chunks = n / maxThreadsThisBlock;
    unsigned m_leftovers = n % maxThreadsThisBlock;
    
    if ((m_chunks == 0) && (m_leftovers > 0)){
      // can't even fill a block
      blocks = dim3(1, 1, 1); 
      threads = dim3((m_leftovers), 1, 1);
    } 
    else {
      // normal case
      if (m_leftovers > 0){
	// not aligned, add an additional block for leftovers
	blocks = dim3(m_chunks + 1, 1, 1);
      }
      else{
	// aligned on block boundary
	blocks = dim3(m_chunks, 1, 1);
      }
      threads = dim3(maxThreadsThisBlock , 1, 1);
    }
    
    radixGlobalSetup_kernel<<<blocks, threads>>>(data_out, a, bit,axis);
    CUT_CHECK_ERROR("Kernel execution failed");

    // Call to CUDPP Scan function
    //cudppScan(d_temp_addr_uint, data_out, n, config);                                Deprecated
    CUDPPResult cudpp_result = cudppScan (planHandle, (void*)d_temp_addr_uint, (void*)data_out, n);

    CUT_CHECK_ERROR("Kernel execution failed");
    
    if (cudpp_result != CUDPP_SUCCESS)
      printf ("Error while performing CUDPP Scan\n");

    radixGlobalSplit_kernel<<<blocks, threads>>>(d_out, d_temp_addr_uint,a, bit,n- 1,axis,nD);
    
    int ns;
    hipMemcpy(&ns,nD,sizeof(unsigned int),hipMemcpyDeviceToHost);
    s = a + ns - 1 ;
    if(bit==0)
      return s;	
    if( ns > k )
      return radixMedianD(a,s,axis,k,bit>>1,data_out,d_temp_addr_uint,d_out,planHandle,nD);
    else
      return radixMedianD(s+1,b,axis,k-ns,bit>>1,data_out,d_temp_addr_uint,d_out,planHandle,nD);
  }
}

void constructD(float3 * NodePos1, float3 * NodePos2, kdNodeInt * n,kdNodeFloat * nf,
		int i,int a,float d,float u,float l,float r, int limit,
		unsigned int * data_out, unsigned int * d_temp_addr_uint, float3 * d_out,CUDPPHandle planHandle, unsigned int * nD,
		complexDevice * OuterD)
{
  static int avail = 2;
  if(i==1)
    avail = 2;
  (n+i-1)->numElems = int(NodePos2-NodePos1)+1;
  dim3 blocks, threads;
  
  unsigned m_chunkSize = maxThreadsThisBlock;
  unsigned m_chunks = (n+i-1)->numElems / m_chunkSize;
  unsigned m_leftovers = (n+i-1)->numElems % m_chunkSize;
  
  if ((m_chunks == 0) && (m_leftovers > 0)){
    // can't even fill a block
    blocks = dim3(1, 1, 1); 
    threads = dim3((m_chunkSize), 1, 1);
  } 
  else{
    // normal case
    if (m_leftovers > 0){
      // not aligned, add an additional block for leftovers
      blocks = dim3(m_chunks + 1, 1, 1);
    }
    else{
      // aligned on block boundary
      blocks = dim3(m_chunks, 1, 1);
    }
    threads = dim3(m_chunkSize, 1, 1);
  }
  
  fcomplex * Outer = (fcomplex *) malloc( 4*(m_chunks+1)*sizeof(fcomplex));
  
  // check if kernel execution generated and error
  CUT_CHECK_ERROR("Kernel execution failed");
  
  complexDevice center; center.x = ((nf+i-1)->center).r; center.y = ((nf+i-1)->center).i;
  hipMalloc((void**)&OuterD, 4*(m_chunks+1)*sizeof(fcomplex));
  calcOuter<<<blocks, threads,4*sizeof(fcomplex)*threads.x>>>(NodePos1,OuterD,(n+i-1)->numElems,center);
  CUT_CHECK_ERROR("Kernel execution failed");
	
  hipMemcpy(Outer, OuterD, 4*(m_chunks+1)*sizeof(fcomplex), hipMemcpyDeviceToHost);
  for(int k = 0; k < m_chunks+1; k+=4)
    for(int j = 0; j < 4; j++)
      (nf+i-1)->Outer[j]= Cadd(Outer[k+j],(nf+i-1)->Outer[j]);
  
  hipFree(OuterD);
  free(Outer);
  
  // check if kernel execution generated and error
  CUT_CHECK_ERROR("Kernel execution failed");
	
  if((n+i-1)->numElems <= limit){
    float3 *start = (float3*)malloc(((n+i-1)->numElems)*sizeof(float3));
    hipMemcpy(start, NodePos1, ((n+i-1)->numElems)*sizeof(float3), hipMemcpyDeviceToHost);
    for(int j = 0; j < (n+i-1)->numElems; j++,start++){
      (n+i-1)->graphNodes[j]= (int)start->z;
    }	
    return;
  }
	
  //CUT_CHECK_ERROR("Kernel execution failed");
  float l1,l2,r1,r2,u1,u2,d1,d2;
        	
  unsigned int bit = 1<<(sizeof(float)*8-1);

  float3 * mid = radixMedianD(NodePos1,NodePos2,a,(int)((NodePos2-NodePos1)/2)+1,bit,data_out,d_temp_addr_uint,d_out,planHandle,nD);
  
  float3 midElem;
  hipError_t err = hipMemcpy(&(midElem), mid, sizeof(float3),hipMemcpyDeviceToHost);	
  CUT_CHECK_ERROR("Kernel execution failed");	
  if(a){
    (nf+i-1)->split = midElem.y;
    l1 =(nf+i-1)->left;  l2=(nf+i-1)->left;
    r1 =(nf+i-1)->right; r2 = (nf+i-1)->right;
    u1 =(nf+i-1)->split; u2 = (nf+i-1)->up;
    d1 =(nf+i-1)->down; d2 = (nf+i-1)->split;
  }
  else{
    (nf+i-1)->split = midElem.x;
    l1 =(nf+i-1)->left; l2=(nf+i-1)->split;
    r1 =(nf+i-1)->split; r2 = (nf+i-1)->right;
    u1 = (nf+i-1)->up; u2 = (nf+i-1)->up;
    d1 = (nf+i-1)->down; d2 = (nf+i-1)->down;
  }
  
  int c = avail++; 
  int c2 = avail++;
  
  kdNodeInitD(n,nf,c,1-a,d1,u1,l1,r1);
  kdNodeInitD(n,nf,c2,1-a,d2,u2,l2,r2);
  
  (n+i-1)->child = c;
  (n+c-1)->next = c2;
  (n+c2-1)->next = (n+i-1)->next;
  
  constructD(NodePos1,mid-1,n,nf,c,1-a,d1,u1,l1,r1,limit,data_out, d_temp_addr_uint, d_out,planHandle,nD,OuterD);
  constructD(mid,NodePos2 ,n,nf,c2,1-a,d2,u2,l2,r2,limit,data_out, d_temp_addr_uint, d_out,planHandle,nD,OuterD);	 
}

#include "hip/hip_runtime.h"
/**************************************************************************************
Copyright (C) Apeksha Godiyal, 2008
Copyright (C) Gerardo Huck, 2009


This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://wwwº.gnu.org/licenses/>.

See license.h for more information.
**************************************************************************************/
#include "license.h"



#include "graph.h"
#include <stdlib.h>
#include <cmath>
#include "common.h"



void initGraph(graph* g, int n)
{
  g->EPSILON = (float)0.000001;
  g->mMaxIterations = 750;
  g->numVertices=n;
  g->screen_width=SCREEN_W;
  g->screen_hieght=SCREEN_H;
  g->currentIteration = 0;
  g->temperature = (float)g->screen_width ;
  g->edge_index=0;
  g->Converged = false;
  float forceConstant = sqrt((float)g->screen_hieght * (float)g->screen_width  /(float) g->numVertices);
  g->attraction_multiplier = 0.75 * forceConstant;
  g->repulsion_multiplier = 0.75 * forceConstant;
  g->coolTimes = 0;
  g->level = 0;
}

bool Converging(graph * g, float * Disp)
{
  for(int i=0;i<g->numVertices;i++){
    if(abs(Disp[2*i])> 0.0 && abs(Disp[2*i+1])>0.0)
      return false;
  }
  return true;
}

bool incrementsAreDone(graph * g) 
{
	if(g->currentIteration>=g->mMaxIterations) 
	  return true;
        return false;
}

void cool(graph * g, int initialNoIterations) 
{
  g->currentIteration++;
  g->temperature *= ((float) 1.0 - g->currentIteration / (float) (g->mMaxIterations));
        
  //Todo: Remove this
  if(g->level > 20){
    if(g->currentIteration == 1)
      g->currentIteration=g->mMaxIterations;	
  }
  else{
    if(g->currentIteration == initialNoIterations/(3*g->level+1))
      g->currentIteration=g->mMaxIterations;	
  }
}	

void calcPositions(int i,float2 * NodePos, float2 * Disp, graph * g) 
{
  float xdisp=Disp[i].x;
  float ydisp=Disp[i].y;
  float deltaLength = max(g->EPSILON, sqrt((xdisp*xdisp)+(ydisp*ydisp)));
  
  float newXDisp = xdisp/ deltaLength ;//* min(deltaLength, g->temperature);
  
  float newYDisp = ydisp / deltaLength ;//* min(deltaLength, g->temperature);
  NodePos[i].x+=newXDisp;
  NodePos[i].y+=newYDisp;
  float borderWidth = g->screen_width / (float)50.0;
  float newXPos = NodePos[i].x;
  if (newXPos < borderWidth) {
    newXPos = borderWidth + (rand()/(float(RAND_MAX)+1)) * borderWidth * (float)2.0;
  } 
  else if (newXPos > (g->screen_width - borderWidth)) {
    newXPos =  g->screen_width- borderWidth - (rand()/(float(RAND_MAX)+1))* borderWidth * (float)2.0;
  }
        
  float newYPos = NodePos[i].y;
  if (newYPos < borderWidth) {
    newYPos = borderWidth + (rand()/(float(RAND_MAX)+1)) * borderWidth * (float)2.0;
  } 
  else if (newYPos > (g->screen_hieght - borderWidth)) {
    newYPos = g->screen_hieght - borderWidth - (rand()/(float(RAND_MAX)+1)) * borderWidth * (float)2.0;
  }
  
  NodePos[i].x=newXPos;
  NodePos[i].y=newYPos;
}

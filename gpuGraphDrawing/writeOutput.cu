/**************************************************************************************
Copyright (C) Apeksha Godiyal, 2008
Copyright (C) Gerardo Huck, 2009


This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.

See license.h for more information.
**************************************************************************************/
#include "license.h"




void writeOutput(graph * g){
	
	float zero = 0;
	float one = 1;
	FILE * oEdges =fopen("data\\4970_edge_6455.oedge","w");
	if(!oEdges) error("cannot open edge file");
	
	for(int i = 0; i < g->numVertices; i++)
		for(int j = g->AdjMatIndex[i]; j < g->AdjMatIndex[i+1]; j++)
			{
			int e2 = g->AdjMatVals[j];
			if(i < e2)
				fprintf(oEdges, "%d %d \r",i,e2);
			}
	fclose(oEdges);
	
	FILE * oVertices =fopen("data\\4970_edge_6455.ocoordinate","w");
	if(!oVertices) error("cannot open coordinate file");
	
	for(int i = 0; i < g->numVertices; i++)
		fprintf(oVertices, "%f %f %f\r",g->NodePos[i].x,g->NodePos[i].y,zero);
		
	fclose(oVertices);
	
	FILE * oGraph =fopen("data\\4970_edge_6455.graph","w");
	if(!oGraph) error("cannot open main.graph file");
	
	fprintf(oGraph, "./data/4970_edge_6455.ocoordinate\n");
	fprintf(oGraph, "./data/4970_edge_6455.oedge\n");
	fprintf(oGraph, "./data/4970_edge_6455.oweight\n");
	
	fclose(oGraph);
	
	FILE * oWeight =fopen("data\\4970_edge_6455.oweight","w");
	if(!oWeight) error("cannot open weight file");
	
	for(int i = 0; i < g->numVertices; i++)
		fprintf(oWeight, "%f\n",one);
		
	fclose(oWeight);
	
}


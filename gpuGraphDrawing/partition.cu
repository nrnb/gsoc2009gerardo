#include "hip/hip_runtime.h"
/**************************************************************************************
Copyright (C) Apeksha Godiyal, 2008
Copyright (C) Gerardo Huck, 2009


This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.

See license.h for more information.
**************************************************************************************/
#include "license.h"




__device__ void setFlagBit(unsigned int * cmpflags, const float3 * idata, 
                           unsigned int bit,int axis)
{
    const int addr0 =  threadIdx.x;
    cmpflags[addr0] = axis?((((unsigned int)(idata[addr0].y) & bit) == 0) ? 1 : 0):((((unsigned int)(idata[addr0].x) & bit) == 0) ? 1 : 0);
}

__global__ void radixGlobalSetup_kernel(unsigned int * g_odata, 
                                        const float3 * g_idata, 
                                        unsigned int bit,
                                        int axis)
{
    const float3 * idata = g_idata + blockIdx.x * blockDim.x;
    unsigned int * odata = g_odata + blockIdx.x * blockDim.x;
    // deposit the correct bit into d_out
    setFlagBit(odata,idata, bit,axis);
}



__global__ void radixGlobalSplit_kernel(float3 * g_data_out, 
                                        unsigned int * g_addr, 
                                        float3 * g_data_in, 
                                        unsigned int bit, unsigned int last, int axis,unsigned int * nD)
{
#ifdef __DEVICE_EMULATION__
    bool debug = false;
#endif
    const int n = blockDim.x;
    const int bid = blockIdx.x;
    const int offset = __mul24(bid, n);
    const unsigned int * addr = g_addr + offset;
    const float3       * data_in = g_data_in + offset;
    const int thid = threadIdx.x;
    const int myid0 = offset + thid;
    
    __shared__ unsigned int totalFalses;
    // g_addr contains output of enumerate
    // g_temp contains input to enumerate (0/1 per element)

#ifdef __DEVICE_EMULATION__
    if (debug)
    {
        __syncthreads();
        if (thid == 0) { printf("last: %d, blockid: %d, myid0: %d, myid1: %d\n", last, bid, myid0, myid1); fflush(stdout); }
        __syncthreads();
    }
#endif

    const int addr0 = (myid0 > last) ? last : thid;
    
    const int g_addr0 = addr0 + offset;
    
    // flag is 0 if false, 1 if true
    unsigned int flag0;
    flag0 = axis? ((((unsigned int)data_in[addr0].y & bit) == 0) ? 1 : 0): ((((unsigned int)data_in[addr0].x & bit) == 0) ? 1 : 0); // store in reg
    if (thid == 0)
    {
        totalFalses = 
             axis? (((((unsigned int)g_data_in[last].y & bit) == 0) ? 1 : 0) + g_addr[last]):(((((unsigned int)g_data_in[last].x & bit) == 0) ? 1 : 0) + g_addr[last]);
		if(bid==0)
			*nD = totalFalses;
    }
    __syncthreads();
        
    // read in addr into laddr (local addr)
    unsigned int laddr0;
    laddr0 = addr[addr0];
        
    // addr is correct for falses (flag == 0), not for trues (flag == 1)
        
       
    if (!flag0)
    {
        laddr0 = g_addr0 - laddr0 + totalFalses;
    }
        
       
        
    // scatter those bad boys into output data array g_data
    // possible problem - running many blocks might mean we have to
    // have different input/output buffers here
    float3 data0;
    data0 = data_in[addr0];
   
    if (myid0 <= last)
    {
        g_data_out[laddr0] = data0;
    }
}


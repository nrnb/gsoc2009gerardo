#include "hip/hip_runtime.h"
/**************************************************************************************
Copyright (C) Apeksha Godiyal, 2008
Copyright (C) Gerardo Huck, 2009


This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.
You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.

See license.h for more information.
**************************************************************************************/
// GPLv3 License
#include "license.h"

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <cmath>
#include <ctime>

// includes, project
#include "cutil.h"
#include "GL/glut.h"
#include "cudpp.h"

// includes, kernels
#include <kernel.cu>

// Include other source files
#include "grap.cu"
#include "kdNode.cu"
#include "pkdNode.cu"
#include "common.h"
		       //#include "readFile.cu"
		       //#include "writeOutput.cu"
#include "scope.h"
#include "display.h"

// This function calculates one step of the force-driven layout process, updating the nodes position
void advancePositions(graph* currentGraph, globalScope *scope)
{
  hipMemcpyToSymbol(HIP_SYMBOL(gd), currentGraph, sizeof(graph));

  // Check if kernel execution generated and error
  CUT_CHECK_ERROR("Kernel execution failed");	
  
  for (int i = 0; i < currentGraph->numVertices; i++){
    scope->NodeTemp[i].x = currentGraph->NodePos[i].x;
    scope->NodeTemp[i].y = currentGraph->NodePos[i].y;
    scope->NodeTemp[i].z = i;
  }
  
  hipMemcpy(scope->a, scope->NodeTemp, currentGraph->numVertices * sizeof(float3), hipMemcpyHostToDevice);
  
  // Configure CUDPP Scan Plan
  CUDPPHandle planHandle;
  cudppPlan (&planHandle, scope->config, currentGraph->numVertices, 1, 0); // rows = 1, rowPitch = 0
  
  int sizeInt   = currentGraph->numVertices * sizeof(kdNodeInt);
  int sizeFloat = currentGraph->numVertices * sizeof(kdNodeFloat);
  
  // Check if the KDTREE has to be rebuilded
  if((currentGraph->currentIteration < 4) ||(currentGraph->currentIteration%20==0) ){

    // Decide whether the KDTREE is goint to be builded in the CPU or in the GPU
    if (currentGraph->numVertices < 50000){ //CPU
      kdNodeInit(scope->rootInt, scope->rootFloat, 1, 0, 0, SCREEN_W,0, SCREEN_H);
      construct(scope->NodeTemp, scope->NodeTemp + currentGraph->numVertices - 1, scope->rootInt, scope->rootFloat, 1, 0, 0, SCREEN_W, 0, SCREEN_H, 3);
    }
    else{                               //GPU   
      kdNodeInitD(scope->rootInt, scope->rootFloat, 1, 0, 0, SCREEN_W, 0, SCREEN_H);
      constructD(scope->a, scope->a + currentGraph->numVertices - 1, scope->rootInt, scope->rootFloat, 1, 0, 0, SCREEN_W, 0, SCREEN_H, 3, scope->data_out, scope->d_temp_addr_uint, scope->d_out, planHandle, scope->nD, scope->OuterD );
    }
  }
  	
  // Copy data to device
  hipMemcpy (scope->NodePosD,   currentGraph->NodePos, (currentGraph->numVertices * sizeof(float2)), hipMemcpyHostToDevice);
  hipMemcpy (scope->treeIntD,   scope->rootInt,        sizeInt,                                      hipMemcpyHostToDevice);
  hipMemcpy (scope->treeFloatD, scope->rootFloat,      sizeFloat,                                    hipMemcpyHostToDevice);

  hipBindTexture (0, texNodePosD, scope->NodePosD,   (sizeof(float2) * currentGraph->numVertices));
  hipBindTexture (0, texInt,      scope->treeIntD,   sizeInt                                     );
  hipBindTexture (0, texFloat,    scope->treeFloatD, sizeFloat                                   );
  
  hipMemcpy(scope->AdjMatIndexD, currentGraph->AdjMatIndex, (currentGraph->numVertices + 1) * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(scope->AdjMatValsD,  currentGraph->AdjMatVals,  (currentGraph->numEdges)        * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(scope->edgeLenD,     currentGraph->edgeLen,     (currentGraph->numEdges)        * sizeof(int), hipMemcpyHostToDevice);

  // Check if kernel execution generated and error
  CUT_CHECK_ERROR("Kernel execution failed");
  
  hipBindTexture (0, texAdjMatValsD, scope->AdjMatValsD, (currentGraph->numEdges) * sizeof(int));
  hipBindTexture (0, texEdgeLenD,    scope->edgeLenD,    (currentGraph->numEdges) * sizeof(int));

  // Check if kernel execution generated and error
  CUT_CHECK_ERROR("Kernel execution failed");
    
  // Execute the kernel, calculate forces
  calculateForces<<< scope->blocks, scope->threads >>>(currentGraph->numVertices, scope->DispD, scope->AdjMatIndexD);
  
  // Check if kernel execution generated and error
  CUT_CHECK_ERROR("Kernel execution failed");

  hipMemcpy(scope->Disp, scope->DispD, currentGraph->numVertices * sizeof(float2), hipMemcpyDeviceToHost);
	
  // Calculate new positions of nodes, based on the force calculations
  for (int i = 0; i < currentGraph->numVertices; i++)
    calcPositions (i, currentGraph->NodePos, scope->Disp, currentGraph); 

  // Decrease the temperature of graph g
  cool(currentGraph, scope->initialNoIterations); 

  // Destroy CUDPP Scan Plan
  cudppDestroyPlan(planHandle);
}


// This function coarses a graph, by obtaining a maximal independant subset of it
graph* coarsen(graph *g, globalScope* scope)
{
  graph *	rg = (graph*) malloc(sizeof(graph));     // New graph which will hold the result of the coarsening
  bool *	used   = (bool*) calloc(g->numVertices,sizeof(bool));
  int *		newNodesNos = (int*) calloc(g->numVertices+1,sizeof(int));
  int		current = 0;
  int		left = g->numVertices;
  int		numParents = 0;

  rg->parent = (int*) calloc (g->numVertices, sizeof(int));
  
  while (left > 0){
    left--;
    newNodesNos[numParents] = current;
    rg->parent[current] = numParents;
    used[current] = 1;
    
    for (int x = g->AdjMatIndex[current]; x < g->AdjMatIndex[current+1]; x++){
      int j = g->AdjMatVals[x];
      if (!used[j])
	left --;
      used[j] = 1;
      rg->parent[j] = numParents;
    }
    numParents++;

    // If there is any node left, search for an unused one
    if (left>0)
      while ((used[current]))
	current++;
  }
  
  free(used);
  
  initGraph(rg,numParents);

  int numEdges = 0;
  rg->NodePos     = (float2 *) malloc((numParents)*sizeof(float2));
  rg->AdjMatIndex =  (int * )  calloc(numParents+1, sizeof(int));
  rg->AdjMatVals  =  (int * )  calloc(g->numEdges,  sizeof(int));
  rg->edgeLen     =  (float * )calloc(g->numEdges,  sizeof(float));
  
  for(int i = 0; i < numParents; i++){
    rg->NodePos[i].x = rand() % SCREEN_W;
    rg->NodePos[i].y = rand() % SCREEN_H;
  }
  
  for ( int i = 0; i < numParents; i++){
    int * usedChild = (int *) calloc(numParents,sizeof(int));
    int node = newNodesNos[i];
    for(int x = g->AdjMatIndex[node]; x < g->AdjMatIndex[node+1]; x++){
      int j = g->AdjMatVals[x];
      if (rg->parent[j] != i)
	usedChild[rg->parent[j]] = 1;
      else{
	for(int y = g->AdjMatIndex[j]; y < g->AdjMatIndex[j+1]; y++){
	  int neighbor = g->AdjMatVals[y];
	  usedChild[rg->parent[neighbor]] = 1;
	}
      }
    }
    
    for ( int k = 0; k < numParents; k++){
      if (usedChild[k]){
	rg->AdjMatVals[numEdges] = k;
	rg->edgeLen[numEdges] = scope->EDGE_LEN;
	numEdges++;
      }
    }
      
    rg->AdjMatIndex[i+1] = numEdges;
    free(usedChild);
  }  
  
  rg->numEdges = numEdges;
  return rg;
}


// This function just applies a one step advance to a graph position
void exactLayoutOnce(globalScope* scope, graph* currentGraph){
  advancePositions(currentGraph, scope);
}

// This funcion initializes a graph position, using the position of nodes in the coarsed graph (if it exists) as a guide
// It also deallocates the memory used by the coarsed graph
void nextLevelInitialization (graph g, graph* coarseGraph, globalScope* scope){
  
  // Nodes that exists in coarseGraph remain in the same position
  for (int i = 0; i < g.numVertices; i++){
    g.NodePos[i].x = coarseGraph->NodePos[coarseGraph->parent[i]].x ;
    g.NodePos[i].y = coarseGraph->NodePos[coarseGraph->parent[i]].y ;
  }
  
  
  for(int j = 0; j < scope->interpolationIterations; j++){
    for(int i = 0; i < g.numVertices; i++){
      int degree = g.AdjMatIndex[i+1] - g.AdjMatIndex[i];
      float2 pi; pi.x=0;pi.y=0;
      for(int k = g.AdjMatIndex[i]; k < g.AdjMatIndex[i+1]; k++){	
	int j = g.AdjMatVals[k];
	pi.x+=g.NodePos[j].x;
	pi.y+=g.NodePos[j].y;
      }
      if(degree){
	g.NodePos[i].x = 0.5 * ( g.NodePos[i].x+ (1.0/degree)*pi.x);
	g.NodePos[i].y = 0.5 * ( g.NodePos[i].y+ (1.0/degree)*pi.y);
      }
    }
  }
  
  free(coarseGraph->NodePos);
  free(coarseGraph->parent);
  free(coarseGraph->AdjMatIndex);
  free(coarseGraph->AdjMatVals);
  free(coarseGraph->edgeLen);
  free(coarseGraph);
}

// This function creates the MIS (Maximal Independent Set) Filtration of a graph
void createCoarseGraphs(graph* g, int level, globalScope* scope)
{
  scope->gArray[level] = g;
  if(g->numVertices <= scope->coarseGraphSize)
    return;
  
  graph *coarseGraph = coarsen(g, scope);
  
  if (g->numVertices < 1.07 * coarseGraph->numVertices )
    return;
  
  if(g->numVertices - coarseGraph->numVertices > 0 )
    createCoarseGraphs(coarseGraph, level + 1, scope);
}



int calculateLayout (globalScope* scope)
{
  
  // Initialize device, using macro defined in "cutil.h"
  CUT_DEVICE_INIT();

  /*    Initializations    */

  // Number of Nodes
  int  numNodes = (scope->g).numVertices;

  // Amount of memory to be used by integers
  int sizeInt = numNodes * sizeof(kdNodeInt);

  // Amount of memory to be used by floats
  int sizeFloat = numNodes * sizeof(kdNodeFloat);
  
  scope->rootInt   = (kdNodeInt*)   calloc(numNodes, sizeof(kdNodeInt)   );
  scope->rootFloat = (kdNodeFloat*) calloc(numNodes, sizeof(kdNodeFloat) );

  hipMalloc ((void**) &(scope->treeIntD),   sizeInt                   );
  hipMalloc ((void**) &(scope->treeFloatD), sizeFloat                 );
  hipMalloc ((void**) &(scope->NodePosD),   numNodes * sizeof(float2) );
  
  // Check if kernel execution generated and error
  CUT_CHECK_ERROR("Kernel execution failed");
  
  // 
  scope->NodeTemp = (float3*) malloc(numNodes * sizeof(float3));
  hipMalloc((void**) &(scope->a), numNodes * sizeof(float3));
  
  scope->Disp = (float2 *) malloc(numNodes * sizeof(float2));

  hipMalloc ((void**) &(scope->DispD),        numNodes                     * sizeof(float2) );
  hipMalloc ((void**) &(scope->AdjMatIndexD), ((scope->g).numVertices + 1) * sizeof(int)    );
  hipMalloc ((void**) &(scope->AdjMatValsD),  (scope->g).numEdges          * sizeof(int)    );
  hipMalloc ((void**) &(scope->edgeLenD),     (scope->g).numEdges          * sizeof(float)  );
  
  // Initialize parameters for config (see CUDPP in cudpp.h)
  (scope->config).algorithm = CUDPP_SCAN;
  (scope->config).op        = CUDPP_ADD;
  (scope->config).datatype  = CUDPP_INT;
  (scope->config).options   = CUDPP_OPTION_FORWARD | CUDPP_OPTION_EXCLUSIVE; 
  
  // Allocate memory in the Device for data used in CUDPP Scan
  hipMalloc((void**) &(scope->data_out),         sizeof(unsigned int) * scope->g.numVertices);
  hipMalloc((void**) &(scope->d_temp_addr_uint), sizeof(unsigned int) * scope->g.numVertices);
  hipMalloc((void**) &(scope->d_out),            sizeof(float3)       * scope->g.numVertices);
  hipMalloc((void**) &(scope->nD),               sizeof(unsigned int)                       );

  /*      END INITIALIZATIONS   */
  

  /*      GRAPH COARSENING      */
  //printf("Coarsening graph...\n");
  
  //clock_t start, end_coarsen,end_layout;
  //double elapsed_layout,elapsed_coarsen;

  //start = clock();
  
  (scope->gArray)[0] = &(scope->g);
  createCoarseGraphs(&(scope->g), 0, scope);
  scope->numLevels = 0;
  while((scope->gArray)[scope->numLevels] != NULL)
    (scope->numLevels)++;
  (scope->gArray)[scope->numLevels - 1]->level = 0;
  
  //end_coarsen = clock();

  //elapsed_coarsen = ((double) (end_coarsen - start)) / CLOCKS_PER_SEC;

  /*      END OF COARSENING      */


  /*      CALCULATE LAYOUTS      */

  //start = clock();
  //printf("Computing layout...\n");
  
  for(int i = 0; i < (scope->numLevels); i++){
    
    // setup execution parameters
    
    unsigned m_chunks    = (scope->gArray)[(scope->numLevels)-i-1]->numVertices / maxThreadsThisBlock;
    unsigned m_leftovers = (scope->gArray)[(scope->numLevels)-i-1]->numVertices % maxThreadsThisBlock;
    
    if ((m_chunks == 0) && (m_leftovers > 0)){
      // can't even fill a block
      scope->blocks  = dim3(1, 1, 1); 
      scope->threads = dim3(m_leftovers, 1, 1);
    } 
    else {
      // normal case
      if (m_leftovers > 0){
	// not aligned, add an additional block for leftovers
	scope->blocks = dim3(m_chunks + 1, 1, 1);
      }
      else{
	// aligned on block boundary
	scope->blocks = dim3(m_chunks, 1, 1);
      }
      scope->threads = dim3(maxThreadsThisBlock , 1, 1);
    }
    
    if(i < (scope->numLevels) - (scope->levelConvergence))
      while(!incrementsAreDone ((scope->gArray)[(scope->numLevels) - i - 1]))
	exactLayoutOnce(scope, (scope->gArray)[(scope->numLevels) - i - 1]);
  
    if((scope->numLevels) - i - 2 >= 0)                  
      nextLevelInitialization(*(scope->gArray)[(scope->numLevels) - i - 2], scope->gArray[(scope->numLevels) - i - 1], scope);
  }

  //end_layout = clock();
  //elapsed_layout = ((double) (end_layout - start)) / CLOCKS_PER_SEC;

  /*       END OF LAYOUT CALCULATION      */
  
  //printf ("Time for coarsening graph: %f\n", elapsed_coarsen);
  //printf ("Time for calculating layout: %f\n", elapsed_layout);
  
  // Release resorces
  hipFree (scope->AdjMatIndexD);
  hipFree (scope->edgeLenD);
  hipFree (scope->AdjMatValsD);
  hipFree (scope->NodePosD);
  hipFree (scope->DispD);
  hipFree (scope->treeIntD);
  hipFree (scope->treeFloatD);
  hipFree (scope->data_out);
  hipFree (scope->d_temp_addr_uint);
  hipFree (scope->d_out);
  hipFree (scope->nD);
  free (scope->NodeTemp);
  free (scope->rootInt);
  free (scope->rootFloat);
  free (scope->Disp);

  // TODO: release gArray[]

  return 0;
}






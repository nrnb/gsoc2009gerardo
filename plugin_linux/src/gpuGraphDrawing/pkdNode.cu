#include "hip/hip_runtime.h"
/**************************************************************************************
Copyright (C) Apeksha Godiyal, 2008
Copyright (C) Gerardo Huck, 2009


This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.

See license.h for more information.
**************************************************************************************/
#include "license.h"

#include "kdNode.h"
#include "cudpp.h"
#include "complexDevice.h"

void kdNodeInit(kdNodeInt * n,kdNodeFloat * nf,int i,unsigned int a,float d,float u,float l,float r){
	int k;
	(n+i-1)->i=i;
	(n+i-1)->numElems=0;
	(n+i-1)->axis=(a);
	(nf+i-1)->up=(u);
	(nf+i-1)->down=(d);
	(nf+i-1)->left=(l);
	(nf+i-1)->right=(r);
	(nf+i-1)->center = Complex((l+r)/2,(u+d)/2); 
	(n+i-1)->child=0;
	(n+i-1)->next=0;
	(nf+i-1)->radius = sqrt(((nf+i-1)->left-(nf+i-1)->center.r)*((nf+i-1)->left-(nf+i-1)->center.r) + ((nf+i-1)->up-(nf+i-1)->center.i)*((nf+i-1)->up-(nf+i-1)->center.i));
	
	for(k = 0; k < 4; k++){
		(nf+i-1)->Outer[k]=Complex(0,0);
	}
}

void print(kdNodeInt * n, kdNodeFloat * nf,int i){
	if(!n)
		return;
	if(!nf)
		return;
	
	if(((n+i-1)->child==0)){
		printf("%d, %f, %f, %f %f %f\n",(n+i-1)->numElems,(nf+i-1)->down,(nf+i-1)->up,(nf+i-1)->left,(nf+i-1)->right,(nf+i-1)->radius);
		if((n+i-1)->next)
			print(n,nf,(n+i-1)->next);
	}
	else{
		if((n+i-1)->child)
			print(n,nf,(n+i-1)->child);
	}
}

void swap(float3 * greater,float3 * less){
	float3 temp = *greater;
	*greater = *less;
	*less    = temp;
}

float3 * radixPartition(float3 * a, float3 * b, unsigned int bit, int axis){
	// y axis
	unsigned int o = 1;
	o = o<<(bit);
	int n = (int)(b-a)+1;
	if(axis){
		int i=0;
		
		while((i<n) && (((unsigned int)(b-i)->y) & o) )
			i++;
		float3 * greater = b-i;
		if(i==n){
			return greater;
		}
		i = 0;
		while((i<n) && !(((unsigned int)(a+i)->y)&o))
			i++;
		float3 * less = a+i; 
		if(i==n){
			return b;
		}
		for(; less < greater; ){
				swap(greater,less);
				greater--;
				less++;
				while(!((unsigned int)(less->y)&o))
					less++;
				while(((unsigned int)(greater->y)&o))
					greater--;
		}
		return greater;
	}
	// x axis
	else{
		int i=0;
		while((i<n) && (((unsigned int)(b-i)->x)&o) )
			i++;
		float3 * greater = b-i;
		if(i==n)
			return greater;
		i = 0;
		while((i<n) && !(((unsigned int)(a+i)->x)&o))
			i++;
		float3 * less = a+i;
		if(i==n)
			return b;
		for(; less < greater; ){
				swap(greater,less);
				greater--;
				less++;
				while(!((unsigned int)(less->x)&o))
					less++;
				while(((unsigned int)(greater->x)&o))
					greater--;
		}
		return greater;
	}
}

float3 * radixMedian(float3 * a, float3 * b, int axis,int k, unsigned int bit){

	float3 * s;
	if((int)bit < 0)
		return (a+(int)((b-a)/2));
	
	if(a==b)
		return a;
	else{
		s = radixPartition(a,b,bit,axis);
		int ns = (int)(s - a) + 1;
		if( ns > k )
			return radixMedian(a,s,axis,k,bit-1);
		else
			return radixMedian(s+1,b,axis,k-ns,bit-1);
	}
}
void construct(float3 * NodePos1, float3 * NodePos2, kdNodeInt * n,kdNodeFloat * nf,
			   int i,int a,float d,float u,float l,float r, int limit){
	
	static int avail = 2;
	if(i==1)
		avail = 2;
	(n+i-1)->numElems = int(NodePos2-NodePos1)+1;
	for(float3 * start = NodePos1; start <= NodePos2; start++){
		fcomplex zi = Complex(start->x,start->y);
		fcomplex z_v_minus_z_0_over_k = Csub(zi,((nf+i-1)->center));
		for(int k = 0; k < 4; k++){
			fcomplex temp = Complex(-1/(float)(k+1),0);
			(nf+i-1)->Outer[k] = Cadd((nf+i-1)->Outer[k], Cmul(z_v_minus_z_0_over_k,temp));
			z_v_minus_z_0_over_k = Cmul(z_v_minus_z_0_over_k,Csub(zi,(nf+i-1)->center));
		}
	}

	if((n+i-1)->numElems <= limit){
		float3 * start = NodePos1;
		for(int j = 0; j < (n+i-1)->numElems; j++,start++)
			(n+i-1)->graphNodes[j]= start->z;
		return;
	}
	
	float l1,l2,r1,r2,u1,u2,d1,d2;
	float3 * mid = radixMedian(NodePos1,NodePos2,a,(int)((NodePos2-NodePos1)/2)+1, sizeof(float)*8-1);
	
	if(a){
		(nf+i-1)->split = mid->y;
		l1 =(nf+i-1)->left;  l2=(nf+i-1)->left;
		r1 =(nf+i-1)->right; r2 = (nf+i-1)->right;
		u1 =(nf+i-1)->split; u2 = (nf+i-1)->up;
		d1 =(nf+i-1)->down; d2 = (nf+i-1)->split;
	}
	else{
		(nf+i-1)->split = mid->x;
		l1 =(nf+i-1)->left; l2=(nf+i-1)->split;
		r1 =(nf+i-1)->split; r2 = (nf+i-1)->right;
		u1 = (nf+i-1)->up; u2 = (nf+i-1)->up;
		d1 = (nf+i-1)->down; d2 = (nf+i-1)->down;
	}
	
	int c = avail++; 
	int c2 = avail++;

	kdNodeInit(n,nf,c,1-a,d1,u1,l1,r1);
	kdNodeInit(n,nf,c2,1-a,d2,u2,l2,r2);

	(n+i-1)->child = c;
	(n+c-1)->next = c2;
	(n+c2-1)->next = (n+i-1)->next;
	
	construct(NodePos1,mid-1,n,nf,c,1-a,d1,u1,l1,r1,limit);
	construct(mid,NodePos2 ,n,nf,c2,1-a,d2,u2,l2,r2,limit);
	 
}
void insertList(kdNodeInt * n,int x,float3 * NodePos,float pos, int index,int limit){
	int i,j;

	if((n+x-1)->axis){
		for( i = 0; (i < (n+x-1)->numElems) && ((NodePos[(n+x-1)->graphNodes[i]].y) <= pos); i++);
		for(j = (n+x-1)->numElems; j > i; j--){
			(n+x-1)->graphNodes[j] = (n+x-1)->graphNodes[j-1];
		}
		(n+x-1)->graphNodes[i] = index;
	}
	else{
		for( i = 0; (i < (n+x-1)->numElems) && ((NodePos[(n+x-1)->graphNodes[i]].x) <= pos); i++);
		for(j = (n+x-1)->numElems; j > i; j--){
			(n+x-1)->graphNodes[j] = (n+x-1)->graphNodes[j-1];
		}
		(n+x-1)->graphNodes[i] = index;
	} 
}

void updateElems(kdNodeInt * n, int i, int* start, int* end){
	int j;
	for(j=0;start <=end; start ++,j++){
		(n+i-1)->graphNodes[j] = *start;
	}
	(n+i-1)->numElems = j;
}

void InsertElem(kdNodeInt * n, kdNodeFloat * nf,int i,float3 * NodePos,float x,float y, int index, int limit){
	static int nIntAvail=2;
	if((i==1)&& (index==0))
		nIntAvail=2;
	int k;
	float pos;

 	fcomplex zi = Complex((x),(y));
	fcomplex z_v_minus_z_0_over_k = Csub(zi,((nf+i-1)->center));
	for(k = 0; k < 4; k++){
		fcomplex temp = Complex(-1/(float)(k+1),0);
		(nf+i-1)->Outer[k] = Cadd((nf+i-1)->Outer[k], Cmul(z_v_minus_z_0_over_k,temp));
		z_v_minus_z_0_over_k = Cmul(z_v_minus_z_0_over_k,Csub(zi,(nf+i-1)->center));
	}

	if((n+i-1)->axis)
		pos = y;
	else
		pos = x;

	if((n+i-1)->child==0){
		insertList((n),i,NodePos,pos,index,limit);
		if((n+i-1)->numElems >= limit){
			float u1,d1,l1,r1,u2,d2,l2,r2;
			int numElems = (n+i-1)->numElems+1;
			int splitting_node = (n+i-1)->graphNodes[((numElems)/2)-1];
			(nf+i-1)->split = ((n+i-1)->axis)? NodePos[splitting_node].y:NodePos[splitting_node].x;
			if((n+i-1)->axis){
				l1 =(nf+i-1)->left;  l2=(nf+i-1)->left;
				r1 =(nf+i-1)->right; r2 = (nf+i-1)->right;
				u1 =(nf+i-1)->split; u2 = (nf+i-1)->up;
				d1 =(nf+i-1)->down; d2 = (nf+i-1)->split;
			}
			else{
				l1 =(nf+i-1)->left; l2=(nf+i-1)->split;
				r1 =(nf+i-1)->split; r2 = (nf+i-1)->right;
				u1 = (nf+i-1)->up; u2 = (nf+i-1)->up;
				d1 = (nf+i-1)->down; d2 = (nf+i-1)->down;
			}
			(n+i-1)->child = nIntAvail; kdNodeInit(n,nf,nIntAvail,1-(n+i-1)->axis,d1,u1,l1,r1);
			nIntAvail++;
			(n+nIntAvail-2)->next = nIntAvail; kdNodeInit(n,nf,nIntAvail,1-(n+i-1)->axis,d2,u2,l2,r2);
			updateElems(n,nIntAvail-1,&((n+i-1)->graphNodes[0]),&((n+i-1)->graphNodes[(numElems/2)-1]));
			updateElems(n,nIntAvail,&((n+i-1)->graphNodes[(numElems/2)]),&((n+i-1)->graphNodes[limit]));
			(n+nIntAvail-1)->next = (n+i-1)->next;
			nIntAvail++;
		}
	}
	else {
		int c = (n+i-1)->child;
		if(pos <= (nf+i-1)->split)
			InsertElem((n),nf,c,NodePos,x,y,index,limit);
		else
			InsertElem((n),nf,(n+c-1)->next,NodePos,x,y,index,limit);
	}
	(n+i-1)->numElems++;
}


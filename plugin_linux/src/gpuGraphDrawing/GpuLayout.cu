#include "hip/hip_runtime.h"
/**************************************************************************************
Copyright (C) Apeksha Godiyal, 2008
Copyright (C) Gerardo Huck, 2009


This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.
You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.

See license.h for more information.
**************************************************************************************/
// GPLv3 License
#include "license.h"

// Header with JNI's function declaration
#include "../GpuLayout_ForceDirected.h" 

// GpuGraphDrawing interface
#include "interface.cu"

//#include <unistd.h>



////////////////////////////////////////////////////////////////////////////////////////
// Implementation of JNI call, it initializes the arguments and calls compute_layout()
////////////////////////////////////////////////////////////////////////////////////////

JNIEXPORT jobjectArray JNICALL Java_GpuLayout_ForceDirected_ComputeGpuLayout (JNIEnv*    env, 
								jobject    thisJ, 
								jintArray  AdjMatIndexJ, 
								jintArray  AdjMatValsJ, 
								jint       coarseGraphSizeJ, 
								jint       interpolationIterationsJ, 
								jint       levelConvergenceJ, 
								jdouble    EDGE_LENJ, 
								jint       initialNoIterationsJ, 
								jdouble    hSizeJ, 
								jdouble    vSizeJ
								)
{

  // Scope which will be used for this computation
  globalScope *scope;

  // Create scope
  scope = globalScopeCreate();

  // Set parameters
  scope->coarseGraphSize          = coarseGraphSizeJ;
  scope->interpolationIterations  = interpolationIterationsJ;
  scope->levelConvergence         = levelConvergenceJ;
  scope->EDGE_LEN                 = EDGE_LENJ;
  scope->initialNoIterations      = initialNoIterationsJ;

  ////////////
  // Set graph
  ////////////
  int numNodes,numEdges;

  // Get numNodes, numEdges
  numNodes = env->GetArrayLength(AdjMatIndexJ) - 1; //AdjMatIndexJ has an extra index for marking the end of AdjMatValsJ
  numEdges = env->GetArrayLength(AdjMatValsJ);

  // Initialize Graph
  initGraph(&(scope->g), numNodes); 

  // Save numEdges
  scope->g.numEdges = numEdges;

  // Allocate memory for NodePos, AdjMatIndex, AdjMatVals, edgeLen
  scope->g.NodePos     = (float2*) malloc (numNodes       * sizeof(float2) );
  scope->g.AdjMatIndex =    (int*) malloc ((numNodes + 1) * sizeof(int)    );
  scope->g.AdjMatVals  =    (int*) malloc (numEdges       * sizeof(int)    );
  scope->g.edgeLen     =  (float*) malloc (numEdges       * sizeof(float)  );

  // Get temporary copies of AdjMatIndex, AdjMatVals
  int *temp_AdjMatIndex = env->GetIntArrayElements(AdjMatIndexJ, NULL);
  int *temp_AdjMatVals  = env->GetIntArrayElements(AdjMatValsJ , NULL);
	
  // Copy temporary copies
  memcpy (scope->g.AdjMatIndex, temp_AdjMatIndex, (numNodes + 1) * sizeof(int));
  memcpy (scope->g.AdjMatVals,  temp_AdjMatVals,  (numEdges)     * sizeof(int));
			
  // Free graph in JVM
  env->ReleaseIntArrayElements(AdjMatIndexJ, temp_AdjMatIndex, 0);
  env->ReleaseIntArrayElements(AdjMatValsJ,  temp_AdjMatVals , 0);

  // Initialize node positions 
  for (int i = 0; i < numNodes; i++){
      scope->g.NodePos[i].x = (int)rand() % scope->g.screen_width;
      scope->g.NodePos[i].y = (int)rand() % scope->g.screen_hieght;
  } 
	 
  // Initialize edge lengths
  for (int i = 0; i < scope->g.AdjMatIndex[numNodes]; i++){
      scope->g.edgeLen[i] = scope->EDGE_LEN;
  }			

		
  // Calculate layout
  calculateLayout (scope);
	      
  // Show results in display
  /*  int argc = 1;
  char **argv;
  char aux[] = "";
  argv[0] = aux;
  showGraph (scope, argc, argv);
  free (argv);
  */

  // Create return object
  jobjectArray result;
  
  // Get the class of float[]
  jclass floatArrCls = env->FindClass("[F");
  if (floatArrCls == NULL) {
    return NULL;
  }

  // Result is an object of type float[][]
  result = env->NewObjectArray(numNodes, floatArrCls, NULL);
  if (result == NULL) {
         return NULL; 
  }
  
  // Allocate memory for each float[] (each one contains the coordinates of a single node) and copy node's position
  for (int i = 0; i < numNodes; i++) {

    // Temporary storage for positions
    float tmp[2];

    // Create a float[]
    jfloatArray temp_float_arr = env->NewFloatArray(2);
    if (temp_float_arr == NULL) {
      return NULL; 
    }

    // Save X and Y positions
    tmp[0] = scope->g.NodePos[i].x * SCREEN_W / hSizeJ;
    tmp[1] = scope->g.NodePos[i].y * SCREEN_H / vSizeJ;

    env->SetFloatArrayRegion(temp_float_arr, 0, 2, tmp);
    env->SetObjectArrayElement(result, i, temp_float_arr);
    env->DeleteLocalRef(temp_float_arr);
  }
	 



  // Release resources
  free (scope->g.NodePos);
  free (scope->g.AdjMatIndex);
  free (scope->g.AdjMatVals);
  free (scope->g.edgeLen);



  return result;
}
  

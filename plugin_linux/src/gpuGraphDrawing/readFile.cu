#include "hip/hip_runtime.h"
/**************************************************************************************
Copyright (C) Apeksha Godiyal, 2008
Copyright (C) Gerardo Huck, 2009


This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.

See license.h for more information.
**************************************************************************************/
#include "license.h"
#include "scope.h"

void error(const char * p, const char * p2="")
{
  printf("%s %s\n",p,p2);
  exit(1);
}
			    

// This function reads a graph from a file (from) stored in a quite strange GML format
// TODO: FIX free() call in readGml() function. It produces a segmentation fault, because array edgeLength overflows

/******************************* FORMAT SAMPLE ***********************
graph [
 Creator "makegml" directed 0 label ""
  node [ id 1 ]
  node [ id 2 ]
  node [ id 3 ]
  node [ id 4 ]
  node [ id 5 ]
  node [ id 6 ]
edge [ source 1 target 3 ]
edge [ source 1 target 4 ]
edge [ source 2 target 2 ]
edge [ source 2 target 3 ]
edge [ source 3 target 1 ]
edge [ source 3 target 2 ]
edge [ source 4 target 5 ]
]
**************************************************************************/


void readGml(globalScope* scope, FILE* from)
{
  int numNodes;
  char string[MAX_REC_LEN]; // used to store data readed from the file temporaly

  printf("Reading nodes!");

  // Skip first 2 lines
  fgets(string, MAX_REC_LEN, from ); // graph [
  fgets(string, MAX_REC_LEN, from ); // Creator "makegml" directed 0 label ""


  /*     READ NODES    */

  // Get the offset (starting position)
  fgets(string, MAX_REC_LEN, from ); 

  int i=0;
  while(string[i] != '[')
    i++;
  int startPos = i; 

  // Get number of nodes
  numNodes = atoi(string+startPos + 4);       // get initial node number
  while( fgets(string, MAX_REC_LEN, from) ){ // get a new line
    int n = atoi(string + startPos + 4);        // read node number
    if(n != numNodes + 1)                       // check whether the new node number is numNodes+1 (if not, we're done here)
      break;
    else  
      numNodes = n;
    //printf("!Node:%ld!\n",ftell (from));		
  }
  printf ("\tnumNodes = %d\n", numNodes);

  /*     FINISH READING NODES      */


  // Get the position in the file in which the edge information starts (secFrom)
  long int secFrom  = ftell (from);         
  secFrom -= (long int)(strlen(string) + 1);

  // Allocate memory for NodePos
  scope->g.NodePos = (float2*) malloc((numNodes) * sizeof(float2));

  // Allocate memory for AdjMatIndex 
  scope->g.AdjMatIndex =  (int*) calloc((numNodes + 1), sizeof(int));


  /*     READ EDGES      */

  scope->g.AdjMatIndex[0]=0;                       // Adjacency list of first node starts in position 0
  int numEdges = 0;                          // Initialize numEdges

  printf("Reading edges!"); 

  // Get the offset (starting position) for edge numbers
  i = 0;
  while(string[i] != '[')
    i++;
  startPos = i;

  // Get the edge source node (e1)
  int e1 = atoi(string + startPos + 9 );

  // Go to the position of edge target node
  i = startPos + 9;
  while(string[i]!= 't')
    i++;

  // Get the edge target node (e2)
  int e2 = atoi(string + i + 6);

  // Increase number of edges, Adj Matrix indexes of e1 and e2
  (scope->g.AdjMatIndex[e1])++;
  (scope->g.AdjMatIndex[e2])++;
  numEdges++;
  
  // Process the rest of the edges
  while(fgets(string, MAX_REC_LEN, from )){
    
    // Check if the file is finishing
    if((string[0]==']') || (string[1]==']'))
      break;

    // Increase number of edges
    numEdges++;

    // Get the edge source node (e1)
    e1 = atoi(string + startPos + 9 );
    i=0;
 
    // Go to the position of edge target node
    i=0;
    i=0;
    while(string[i]!= 't')
      i++;

    // Get the edge target node (e2)
    int e2 = atoi(string + i + 6);
    (scope->g.AdjMatIndex[e1])++;
    (scope->g.AdjMatIndex[e2])++;
  }
  
  // Update AdjMatIndex so that each position points to the appropiate element in AdjMatVals
  for(int i = 0; i < numNodes; i++)
    scope->g.AdjMatIndex[i+1] += scope->g.AdjMatIndex[i];
  
  printf("\tNumber of Edges = %d\n",numEdges);

  // Allocate memory for AdjMatVals, edgeLen
  scope->g.AdjMatVals  = (int*) malloc(2 * numEdges * sizeof(int));
  scope->g.edgeLen     = (int*) malloc(2 * numEdges * sizeof(int));

  // Allocate memory for temp, an auxiliary array, initialize it whith zeros
  int *temp      = (int*) calloc(numNodes, sizeof(int));

  // Initialize Graph
  initGraph(&(scope->g), numNodes); 

  // Save numEdges
  scope->g.numEdges = 2*numEdges;

  // Go to secFrom position in file "from" (where the edge information starts)
  fseek ( from, secFrom, SEEK_SET );

  while( fgets(string, MAX_REC_LEN,from )){
    
    // Check if the file is finishing
    if( (string[0]==']') || (string[1]==']') )
      break;

    // Get the edge source node (e1)
    e1 = atoi(string+startPos+9 );

    // Go to the position of edge target node
    i=0;
    while(string[i]!= 't')
      i++;

    // Get the edge target node (e2)
    int e2 = atoi(string + i + 6);

    // Add e1 to adjacency list of e2 and vice versa.
    scope->g.AdjMatVals[scope->g.AdjMatIndex[e1-1]+temp[e1-1]] = e2-1;
    scope->g.AdjMatVals[scope->g.AdjMatIndex[e2-1]+temp[e2-1]] = e1-1;

    // Save edge lenght for this edge
    scope->g.edgeLen[scope->g.AdjMatIndex[e1-1]+temp[e1-1]] = scope->EDGE_LEN;
    scope->g.edgeLen[scope->g.AdjMatIndex[e2-1]+temp[e2-1]] = scope->EDGE_LEN;

    // Increase the number of neighbors already processed of e1 and e2 
    (temp[e1 - 1])++;
    (temp[e2 - 1])++;

  }
 
  // FIX IT! The following free() call gives segmentation fault
  //  free ((void*) temp);
}





/******************************* FORMAT SAMPLE ***********************
7 10
2 3
1 3 7
1 2 6 7
6
6 7
3 4 5 7
2 3 5 6
**********************************************************************/
//In more detail, there are 7 nodes and 10 edges in the graph; node 1 is adjacent to 2,3; node 2 is adjacent to 1,3,7; etc


void readChaco(globalScope* scope, FILE* from)
{
  int numNodes,numEdges;
  char string[MAX_REC_LEN];        // Temporary string in which each line of the file will be temporary stored
  int index = 0;
  int nEdges = 0;
  
  if(!fscanf(from,"%d",&numNodes))
    error("Cannot read 1st file");
  if(!fscanf(from,"%d",&numEdges))
    error("Cannot read 1st file");

  printf ("Number of nodes: %d\n", numNodes);
  printf ("Number of edges: %d\n", numEdges);
  printf ("Reading nodes!!\n");		
  printf ("Reading edges!!\n");

  // Initialize Graph
  initGraph(&(scope->g), numNodes); 

  // Save numEdges
  scope->g.numEdges = 2 * numEdges;

  // Allocate memory for NodePos, AdjMatIndex, AdjMatVals, edgeLen
  scope->g.NodePos     = (float2*) malloc (numNodes       * sizeof(float2) );
  scope->g.AdjMatIndex =    (int*) malloc ((numNodes + 1) * sizeof(int)    );
  scope->g.AdjMatVals  =    (int*) malloc (2 * numEdges   * sizeof(int)    );
  scope->g.edgeLen     =    (int*) malloc (2 * numEdges   * sizeof(int)    );
	
  // First node's adjacency list starts at position 0
  scope->g.AdjMatIndex[0]=0;

  // Read rest of file
  while(fgets(string, MAX_REC_LEN,from )){
    
    if((string[0]==10) || (string[0]==8) ) 
      continue;
    
    // Initialize node position
    scope->g.NodePos[index].x= (int)rand() % scope->g.screen_width;
    scope->g.NodePos[index].y= (int)rand() % scope->g.screen_hieght;
    
    if(scope->g.NodePos[index].x < 0){
      exit(0);
    }
    
    char * first = string;
    int sl=strlen(string);
    
    for(int i=0; i < sl; i++){
    
      if(string[i]==10){
	string[i]='\0';
	int n = atoi(first);
	first = &string[i]; 
	scope->g.AdjMatVals[nEdges] = n - 1;
	scope->g.edgeLen[nEdges] = scope->EDGE_LEN;
	nEdges++;
	break;
      }
      
      if(i == 0 && string[0] == ' '){
	first = string + 1;
	continue;
      }
      if(string[i] != ' ') continue;
      
      while(string[i] == ' ') 
	i++;
      
      string[i-1] = '\0';
      
      if(strlen(first)){
	int n = atoi(first);
	first = &string[i]; 
	scope->g.AdjMatVals[nEdges] = n - 1;
	scope->g.edgeLen[nEdges] = scope->EDGE_LEN;
	nEdges++;
      }
    }
    
    scope->g.AdjMatIndex[index+1] = nEdges;
    index++;
    
  }	
}


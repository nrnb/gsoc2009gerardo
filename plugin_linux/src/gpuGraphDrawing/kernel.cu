#include "hip/hip_runtime.h"
/**************************************************************************************
Copyright (C) Apeksha Godiyal, 2008
Copyright (C) Gerardo Huck, 2009


This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.

See license.h for more information.
**************************************************************************************/
#include "license.h"




#ifndef _HELLOWORLD_H_
#define _HELLOWORLD_H_

#include <stdio.h>
#include "complexDevice.h"
#include "graph.h"
#include "common.h"

enum intMembers{axis = 0, child=1, next=2, numElems=3, graphNodes,i=9};
enum floatMembers{radius=0,center=1,split=3,up=4,down=5,left=6,right=7,Outer=8};

texture<int> texInt;
texture<int> texAdjMatD;
texture<float> texFloat;
texture<int> texAdjMatValsD;
texture<float> texEdgeLenD;
texture<float2> texNodePosD;

__constant__ graph gd[1];
int nIntAvail=2;

__device__ float f_rep_scalar (float d)
	{
		if (d > 0)
		{
			return 1/d;
		}
		else
		{
			//printf("Error FruchtermanReingold:: f_rep_scalar nodes at same position\n");
			return 0;
		}
	}

__device__ void calcExactRepulsion(float3 NodePosS, float2* DispS, int j){
 int i;
 float scalar;          
 for(i = 0; i<tex1Dfetch(texInt,__umul24(j,10)+numElems); i++)
 {
	 if(NodePosS.z != tex1Dfetch(texInt,__umul24(j,10)+graphNodes+i)){
		int k = (tex1Dfetch(texInt,__umul24(j,10)+graphNodes+i));
		float2 NodePos = tex1Dfetch(texNodePosD,k);
		float xDelta = NodePosS.x- NodePos.x;
		float yDelta = NodePosS.y- NodePos.y;
 		float deltaLength = max(gd[0].EPSILON, sqrtf((xDelta * xDelta) + (yDelta * yDelta)));
		scalar = f_rep_scalar(deltaLength)/deltaLength;
		float f_rep_x = scalar * xDelta;
		float f_rep_y = scalar * yDelta;
		
		DispS->x+= f_rep_x;
		DispS->y+= f_rep_y;
	 }
 }
}

__device__ void calcFMMRepulsion(complexDevice z, float dist, float2* Disp, int j){
	complexDevice one; Complex(1,0,&one);
	complexDevice numElements; Complex((float)(tex1Dfetch(texInt,__umul24(j,10)+numElems)),0,&numElements);
	complexDevice z_v_minus_z_0_over_minus_k_minus_1; Cdiv(one,(z),&z_v_minus_z_0_over_minus_k_minus_1);
    complexDevice sum; Cmul(numElements,z_v_minus_z_0_over_minus_k_minus_1,&sum);
          
	  for(int k=0;k<4;k++)
	    {
              Cdiv(z_v_minus_z_0_over_minus_k_minus_1,z,&z_v_minus_z_0_over_minus_k_minus_1);
              complexDevice O; Complex(tex1Dfetch(texFloat,__umul24(j,16)+Outer+2*k),tex1Dfetch(texFloat,__umul24(j,16)+Outer+2*k+1),&O);
			  complexDevice temp1; Complex((float)k+1,0,&temp1);
			  complexDevice temp2; Cmul(O,z_v_minus_z_0_over_minus_k_minus_1,&temp2);
			  complexDevice Outerk;Cmul((temp1),(temp2),&Outerk);
              Csub(sum,Outerk,&sum);
	    }
		
	  Disp->x+=sum.x;
	  Disp->y+=-1*sum.y;
}

__device__ void calcRepulsion(float3 NodePos, float2* Disp){
	int j = 1;
	complexDevice z; Complex(NodePos.x,NodePos.y,&z);
	while(j){
		j--;
		complexDevice Center; Complex(tex1Dfetch(texFloat,__umul24(j,16)+center),tex1Dfetch(texFloat,__umul24(j,16)+center+1),&Center);
		complexDevice z_z0; Csub(z,Center,&z_z0);
		float dist = sqrt(z_z0.x*z_z0.x+z_z0.y*z_z0.y);
		float rad = tex1Dfetch(texFloat,__umul24(j,16)+radius); 
		if( dist > rad){
			calcFMMRepulsion(z_z0,dist,Disp,j);
			j = tex1Dfetch(texInt,__umul24(j,10)+next);
	}
	else{
		if(tex1Dfetch(texInt,__umul24(j,10)+child)==0){
			(calcExactRepulsion(NodePos,Disp,j));
			j = tex1Dfetch(texInt,__umul24(j,10)+next);
		}
		else{
			if(tex1Dfetch(texInt,__umul24(j,10)+child))
				j = tex1Dfetch(texInt,__umul24(j,10)+child);
			else
				j = tex1Dfetch(texInt,__umul24(j,10)+next);
			}
		}
	}
}

__device__ float f_attr_scalar (float d,float ind_ideal_edge_length)
{
  float s;
  float c =  log(d/ind_ideal_edge_length)/log(2.0);
      if (d > 0) 
	s =  c * d * d /
   	    (ind_ideal_edge_length * ind_ideal_edge_length * ind_ideal_edge_length); 
      else 
	s = -1e10;   
 return s;
} 

__device__ void calcAttraction(int j, float2* DispS, float3 NodePosS, float edgeLength) {
	float scalar;
	float xDelta, yDelta;
	float2 NodePos = tex1Dfetch(texNodePosD,j);
    xDelta = NodePosS.x - NodePos.x;
    yDelta = NodePosS.y - NodePos.y;
 
    float deltaLength = max(gd[0].EPSILON, sqrtf((xDelta * xDelta)+ (yDelta * yDelta)));
    
    scalar = f_attr_scalar(deltaLength,edgeLength)/deltaLength;
	float dx = scalar * xDelta;
	float dy = scalar * yDelta; 
        
    DispS->x-=2*dx;
	DispS->y-=2*dy;
}

__global__ void calculateForces(int numNodes, float2 * DispD, int * AdjMatIndexD)
{       
		const int globalId =  __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
        
        if(globalId < numNodes){
			float2 NodePosTemp; 
			float2 Disp= make_float2(0,0); 
			float3 NodePos;
			
			NodePosTemp = tex1Dfetch(texNodePosD,globalId);
			NodePos.x = NodePosTemp.x;
			NodePos.y = NodePosTemp.y;
			NodePos.z = globalId;
			calcRepulsion(NodePos,&Disp);
			int end = AdjMatIndexD[globalId+1]; 
			for(int i = AdjMatIndexD[globalId]; i< end; i++)
					calcAttraction(tex1Dfetch(texAdjMatValsD,i),&Disp,NodePos,tex1Dfetch(texEdgeLenD,i));
			DispD[globalId] = Disp;
		}	
}

__global__ void 
calcOuter(float3 * NodePos, complexDevice * OuterD, int numNodes,complexDevice c){
	extern __shared__ complexDevice OuterS[];
	float3 * idata = NodePos + blockIdx.x * blockDim.x;
	unsigned int thid  = threadIdx.x;
	unsigned int gid   = blockIdx.x * blockDim.x + thid;
	
	complexDevice zi;  
	if(gid < numNodes)
		Complex(idata[thid].x,idata[thid].y,&zi);
	else
		Complex(0,0,&zi);
		
	complexDevice z_v_minus_z_0_over_k; Csub(zi,c,&z_v_minus_z_0_over_k);
	complexDevice temp = z_v_minus_z_0_over_k;
	for(int k = 0; k < 4; k++){
		if(gid < numNodes){
			complexDevice temp2; Complex(-1/(float)(k+1),0,&temp2);
			Cmul(z_v_minus_z_0_over_k,temp2,&OuterS[thid+blockDim.x*k] );
		}
		else
			Complex(0,0,&OuterS[thid+blockDim.x*k]);
		Cmul(z_v_minus_z_0_over_k,temp,&z_v_minus_z_0_over_k);
	}
	
	int offset = 1;
	for (int d = blockDim.x>>1; d > 0; d >>= 1) // build sum in place up the tree
	{
		__syncthreads();
		if (thid < d)
		{
			int ai = offset*(2*thid+1)-1;
			int bi = offset*(2*thid+2)-1;
			Cadd(OuterS[bi],OuterS[ai],&OuterS[bi]);
			ai = ai+blockDim.x;
			bi = bi+blockDim.x;
			Cadd(OuterS[bi],OuterS[ai],&OuterS[bi]);
			ai = ai+blockDim.x;
			bi = bi+blockDim.x;
			Cadd(OuterS[bi],OuterS[ai],&OuterS[bi]);
			ai = ai+blockDim.x;
			bi = bi+blockDim.x;
			Cadd(OuterS[bi],OuterS[ai],&OuterS[bi]);
		}
			offset *= 2;
	}
	if(thid<4){
		OuterD[blockIdx.x*4+thid]=OuterS[(blockDim.x-1)+thid*blockDim.x];
	}
}

#endif
